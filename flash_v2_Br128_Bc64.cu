#include "hip/hip_runtime.h"
#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define WARP_SIZE 32

#define INT4(value) (reinterpret_cast<int4 *>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4 *>(&(value))[0])
#define FLOAT2(value) (reinterpret_cast<float2 *>(&(value))[0])

// Warp Reduce Max
template <const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ float warp_reduce_max_f32(float val) {
#pragma unroll
  for (int mask = kWarpSize >> 1; mask >= 1; mask >>= 1) {
    val = fmaxf(val, __shfl_xor_sync(0xffffffff, val, mask));
  }
  return val;
}

// Warp Reduce Sum
template <const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ float warp_reduce_sum_f32(float val) {
#pragma unroll
  for (int mask = kWarpSize >> 1; mask >= 1; mask >>= 1) {
    val += __shfl_xor_sync(0xffffffff, val, mask);
  }
  return val;
}

__global__
void flash_attention_2_forward_kernel(
    float* Q,
    float* K,
    float* V,
    const int N,
    const int d,
    const int Tc,
    const int Tr,
    const int Bc,
    const int Br,
    const float softmax_scale,
    // float* L,
    float* O
) {
    int tx = threadIdx.x;
    int bx = blockIdx.x; int by = blockIdx.y;  // batch and head index

    int block_width = 8;
    int row = tx / block_width; // row from 0 to 128
    int col = tx % block_width; // col from 0 to 7

    // Offset into Q,K,V,O - different for each batch and head
    int qkv_offset = (bx * gridDim.y * N * d) + (by * N * d);  // gridDim.y = nh
    int lm_offset = (bx * gridDim.y * N) + (by * N);  // offset for L

    // Define SRAM for Q,K,V,S
    extern __shared__ float sram[];
    int kv_tile_size = Bc * d;  // size of Kj, Vj
    int q_tile_size = Br * d;  // size of Qi
    float* Qi = sram;
    float* KVj = &sram[q_tile_size];
    // float* Vj = &sram[q_tile_size + kv_tile_size];
    float* S = &sram[q_tile_size + kv_tile_size];

    for (int i = 0; i < Tr; ++i) {
        // if (i * Br + row >= N)
        //     break;  // break if we are done with the sequence

        // Load Qi from HBM to SRAM, l and m to registers
        // TODO: Add vectorized loading from DRAM //
        // for (int x = 0; x < 4; x++) {
        //     Qi[tx + Bc * Bc * x] = Q[qkv_offset + (q_tile_size * i) + tx + Bc * Bc * x];
        // }
        FLOAT4(Qi[row * d + col * 8]) = FLOAT4(Q[qkv_offset + (q_tile_size * i) + row * d + col * 8]);
        FLOAT4(Qi[row * d + col * 8 + 4]) = FLOAT4(Q[qkv_offset + (q_tile_size * i) + row * d + col * 8 + 4]);

        float row_m_prev = -INFINITY;
        float row_l_prev = 0;

        // Causal mask: j <= i
        for (int j = 0; j < Tc; ++j) {
            int new_row = tx / 16; // row from 0 to 64
            int new_col = tx % 16; // col from 0 to 15
            __syncthreads();
            // Load Kj Vj from HBM to SRAM
            FLOAT4(KVj[new_row * d + new_col * 4]) = FLOAT4(K[qkv_offset + (kv_tile_size * j) + new_row * d + new_col * 4]);
            // FLOAT4(Vj[row * d + col * 4]) = FLOAT4(V[qkv_offset + (kv_tile_size * j) + row * d + col * 4]);
            __syncthreads();
            // S_i^j = softmax_scale * QiKj^T
            // S_i^j[tx][y] = softmax_scale * Sum_{x = 0}^{d-1} Qi[tx][x] * Kj[y][x]
            float row_m = -INFINITY;
            for (int y = 0; y < Bc/block_width; y++) {
                //if (j * Bc + y >= N)
                //    break;  // break if we are done with the sequence
                //if (i * Br + tx < j * Bc + y)
                //    break;
                int col_offset = y * block_width;
                float val = 0;
                for (int x = 0; x < d; x++)
                    val += Qi[(row * d) + x] * KVj[((col+col_offset) * d) + x];
                val *= softmax_scale;
                S[(row * Bc) + col + col_offset ] = val;
                // Find the maximum value in the row S_i^j
                float warp_m =  warp_reduce_max_f32<WARP_SIZE/4>(val);
                row_m = fmaxf(row_m, warp_m);
            }

            __syncthreads();
            // FLOAT4(KVj[row * d + col * 4]) = FLOAT4(K[qkv_offset + (kv_tile_size * j) + row * d + col * 4]);
            FLOAT4(KVj[new_row * d + new_col * 4]) = FLOAT4(V[qkv_offset + (kv_tile_size * j) + new_row * d + new_col * 4]);
            __syncthreads();

            // m_i^j = max(m_i^j-1, row_max(S_i^j))
            float new_row_m = fmaxf(row_m_prev, row_m);

            // P_i^j = exp(S_i^j - m_i^j)
            // P_i^j[tx][y] = exp(S_i^j[tx][y] - m_i^j)
            float row_l = 0;
            for (int y = 0; y < Bc/block_width; y++) {
                //if (j * Bc + y >= N)
                //    break;  // break if we are done with the sequence
                //if (i * Br + tx < j * Bc + y)
                //    break;
                int col_offset = y * block_width;
                float exp_val = __expf(S[(Bc * row) + col + col_offset] - new_row_m);
                S[(Bc * row) + col + col_offset] = exp_val;
                // Sum over P_i^j to get row_sum(P_i^j)
                row_l += warp_reduce_sum_f32<WARP_SIZE/4>(exp_val);
            }

            // l_i^j = (exp(m_i^j-1 - m_i^j) * l_i^j-1) + row_sum(P_i^j)
            float row_m_exp = __expf(row_m_prev - new_row_m);
            float new_row_l = (row_m_exp * row_l_prev) + row_l;

            // O_i^j = diag(exp(m_i^j-1 - m_i^j))^-1 * O_i^j-1 + P_i^jVj
            for (int y = 0; y < d/block_width; y++) {
                float pv = 0;  // Pij * Vj
                int col_offset = y * block_width;
                for (int x = 0; x < Bc; x++) {
                    pv += S[(Bc * row) + x] *KVj[(x * d) + col + col_offset];
                }

                O[qkv_offset + (q_tile_size * i) + (row * d) + col + col_offset] = \
                    row_m_exp * O[qkv_offset + (q_tile_size * i) + (row * d) + col + col_offset] + pv;
            }

            // Update m and l
            row_m_prev = new_row_m;
            row_l_prev = new_row_l;
        }

        // O_i = diag(l_i^{Tc})^-1 * O_i^{Tc}
        for (int y = 0; y < d/block_width; y++) {
            //if (i * Br + tx < y * Bc)
            //    break;
            int col_offset = y * block_width;
            O[qkv_offset + (q_tile_size * i) + (row * d) + col + col_offset] /= row_l_prev;
        }
        // L_i = m_i^{Tc} + log(l_i^{Tc})
        // L[lm_offset + (Br * i) + tx] = row_m_prev + __logf(row_l_prev);
    }
}


torch::Tensor forward(torch::Tensor Q, torch::Tensor K, torch::Tensor V) {
    // TODO: determine Bc, Br dynamically
    const int Bc = 64; const int Br = 128;

    const int B = Q.size(0); const int nh = Q.size(1);
    const int N = Q.size(2); const int d = Q.size(3);

    const int Tc = ceil((float) N / Bc); const int Tr = ceil((float) N / Br);
    const float softmax_scale = 1.0 / sqrt(d);

    // Initialize O, L to HBM
    auto O = torch::zeros_like(Q);
    // auto L = torch::zeros({B, nh, N});
    torch::Device device(torch::kCUDA);
    // L = L.to(device);
    O = O.to(device);

    // Calculate SRAM size needed per block
    int col_tile_size = Bc * d;  // size of Kj, Vj
    int row_tile_size = Br * d;  // size of Qi
    const int sram_size =
          (col_tile_size * sizeof(float))  // SRAM size for Kj, Vj
        + (row_tile_size * sizeof(float))  // SRAM size for Qi
        + (Bc * Br * sizeof(float));  // SRAM size for S

    //hipFuncSetAttribute(reinterpret_cast<const void*>(flash_attention_2_forward_kernel), hipFuncAttributePreferredSharedMemoryCarveout, cudaSharedmemCarveoutMaxShared);
    hipFuncSetAttribute(reinterpret_cast<const void*>(flash_attention_2_forward_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, sram_size);

    int max_sram_size;
    hipDeviceGetAttribute(&max_sram_size, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
    printf("Max shared memory: %d, requested shared memory: %d \n", max_sram_size, sram_size);

    dim3 grid_dim(B, nh);  // batch_size x num_heads
    dim3 block_dim(1024);  // Br x Br (1024) threads per block

    flash_attention_2_forward_kernel<<<grid_dim, block_dim, sram_size>>>(
        Q.data_ptr<float>(), K.data_ptr<float>(), V.data_ptr<float>(),
        N, d, Tc, Tr, Bc, Br, softmax_scale,
        O.data_ptr<float>()
    );
    return O;
}